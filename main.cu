/**
 * Copyright (c) 2020 Nina Herrmann
 *
 * This software is released under the MIT License.
 * https://opensource.org/licenses/MIT
 */

#include <hip/hip_runtime.h>
#include <algorithm>
#include <string>
#include <iostream>
#include <sstream>
#include <fstream>
//#include <mpi.h>
#define MAX_ITER 1000
#ifdef __HIPCC__
#define POW(a, b)      powf(a, b)
#define EXP(a)      exp(a)
#else
#define POW(a, b)      std::pow(a, b)
#define EXP(a)      std::exp(a)
#endif
int rows, cols;
int* input_image_int;
char* input_image_char;
bool ascii = false;
int DEFAULT_TILE_WIDTH = 16;
bool DEBUG = true;
int stencil_size = 2;
#define gpuErrchk(ans)                                                         \
{ gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
                line);
        if (abort)
            exit(code);
    }
}
int readPGM(const std::string& filename, int& rows, int& cols, int& max_color)
{
    std::ifstream ifs(filename, std::ios::binary);
    if (!ifs) {
        std::cout << "Error: Cannot open image file " << filename << "!" << std::endl;
        return 1;
    }
    // Read magic number.
    std::string magic;
    getline(ifs, magic);
    if (magic.compare("P5")) { // P5 is magic number for pgm binary format.
        if (magic.compare("P2")) { // P2 is magic number for pgm ascii format.
            std::cout << "Error: Image not in PGM format!" << std::endl;
            return 1;
        }
        ascii = true;
    }

    // Skip comments
    std::string inputLine;
    while (true) {
        getline(ifs, inputLine);
        if (inputLine[0] != '#') break;
    }

    // Read image size and max color.
    std::stringstream(inputLine) >> cols >> rows;
    getline(ifs, inputLine);
    std::stringstream(inputLine) >> max_color;
    std::cout << "\nmax_color: " << max_color << "\t cols: " << cols << "\t rows: " << rows << std::endl;

    // Read image.
    if (ascii) {
        input_image_int = new int[rows*cols];
        int i = 0;
        while (getline(ifs, inputLine)) {
            std::stringstream(inputLine) >> input_image_int[i++];
        }
    } else {
        input_image_char = new char[rows*cols];
        ifs.read(input_image_char, rows*cols);
    }
    return 0;
}

int writePGM(const std::string& filename, int *out_image, int rows, int cols, int max_color)
{
    std::ofstream ofs(filename, std::ios::binary);
    if (!ofs) {
        std::cout << "Error: Cannot open image file " << filename << "!" << std::endl;
        return 1;
    }

    // Gather full image
    int** img = new int*[rows];
    for (int i = 0; i < rows; i++)
        img[i] = new int[cols];

    // Write image header
    ofs << "P5\n" << cols << " " << rows << " " << std::endl << max_color << std::endl;

    // Write image
    for (int x = 0; x < rows; x++) {
        for (int y = 0; y < cols; y++) {
            unsigned char intensity = static_cast<unsigned char> (out_image[x*cols + y]);
            ofs << intensity;
        }
    }

    if (ofs.fail()) {
        std::cout << "Cannot write file " << filename << "!" << std::endl;
        return 1;
    }

    return 0;
}

__device__
void printsm(int global_col, int global_row, int tile_width, int * data){
    if (global_col == 0 && global_row == 0) {
        printf("[");
        for(int i = 0; i < tile_width; i++){
            printf("\n");
            for(int j = 0; j < tile_width; j++){
                printf("%d;", data[j + i * tile_width]);
            }
        }
        printf("]\n");
    }
}
__device__
void printgm(int global_col, int global_row, int tile_width, int cols, int * data){
    if (global_col == 0 && global_row == 0) {
        printf("[");
        for(int i = 0; i < tile_width; i++){
            printf("\n");
            for(int j = 0; j < tile_width; j++){
                printf("%d;", data[j + i * cols]);
            }
        }
        printf("]\n");
    }
}

__global__
void calcGaussian(const int *input, int *output, int cols, int kw) {
    size_t thread = threadIdx.x + blockIdx.x * blockDim.x;

    int row = thread / cols;
    int col = thread % cols;
    int offset = kw/2;
    float weight = 1.0f;
    float sigma = 1;
    float mean = (float)kw/2;
    //printgm(row, col, 32, cols+kw, input);
    float sum = 0;
    for (int r = 0; r <= kw; ++r) {
        for (int c = 0; c <= kw; ++c) {
            sum += input[(row + r) * cols + col + c] *
                    EXP(-0.5 * (POW((r-mean)/sigma, 2.0) + POW((c-mean)/sigma,2.0))) / (2 * M_PI * sigma * sigma);
        }
    }
    output[row*cols + col] = (int)sum/weight;
}

__global__
void calcGaussianSM(const int *input, int *output, int cols, int rows, int kw, int tile_width) {

    int global_col = blockIdx.y * blockDim.y + threadIdx.y;
    int global_row = blockIdx.x * blockDim.x + threadIdx.x;
    int local_col = threadIdx.y;
    int local_row = threadIdx.x;
    int offset = kw/2;
    float weight = 1.0f;
    float sigma = 1;
    int new_tile_width =  tile_width + kw;
    float mean = (float)kw/2;
    extern __shared__ int data[];

    for (int r = 0; r <= kw; ++r) {
        for (int c = 0; c <= kw; ++c) {
            data[(local_row + r) * new_tile_width + (local_col + c)] = input[((global_row + r) * (cols+kw)) + (global_col + c)];
        }
    }
    __syncthreads();
    //printsm(global_row, global_col, new_tile_width, data);
    float sum = 0;
    for (int r = 0; r <= kw; ++r) {
        for (int c = 0; c <= kw; ++c) {
           sum += data[(local_row + r) * new_tile_width + (local_col + c)] *
                    EXP(-0.5 * (POW((r-mean)/sigma, 2.0) + POW((c-mean)/sigma,2.0))) / (2 * M_PI * sigma * sigma);
        }
    }
    output[(global_col + offset) + global_row * (cols+kw) + (offset * (cols+kw))] = (int)sum/weight;
}

__global__
void calcGaussian_fixed_SM(const int *input, int *output, int cols, int rows, int kw, int tile_width) {

    int global_col = blockIdx.y * blockDim.y + threadIdx.y;
    int global_row = blockIdx.x * blockDim.x + threadIdx.x;
    int local_col = threadIdx.y;
    int local_row = threadIdx.x;
    int offset = kw/2;
    float weight = 1.0f;
    float sigma = 1;
    int new_tile_width =  tile_width + kw;
    float mean = (float)kw/2;
    __shared__ int data[1764];
    for (int r = 0; r <= kw; ++r) {
        for (int c = 0; c <= kw; ++c) {
            data[(local_row + r) * new_tile_width + (local_col + c)] = input[((global_row + r) * (cols+kw)) + (global_col + c)];
        }
    }
    __syncthreads();
    //printsm(global_row, global_col, new_tile_width, data);
    float sum = 0;
    for (int r = 0; r <= kw; ++r) {
        for (int c = 0; c <= kw; ++c) {
            sum += data[(local_row + r) * new_tile_width + (local_col + c)] *
                    EXP(-0.5 * (POW((r-mean)/sigma, 2.0) + POW((c-mean)/sigma,2.0))) / (2 * M_PI * sigma * sigma);
        }
    }
    output[(global_col + offset) + global_row * (cols+kw) + (offset * (cols+kw))] = (int)sum/weight;
}

int testGaussian(std::string in_file, std::string out_file, bool output, int tile_width, int iterations, int iterations_used, std::string file, bool shared_mem, int kw) {
    int max_color;
    hipEvent_t initstart, initstop;
    hipEventCreate(&initstart);
    hipEventCreate(&initstop);
    hipEventRecord(initstart);

    // Read image
    readPGM(in_file, rows, cols, max_color);
    const unsigned int elements = (rows + kw) * (cols + kw);
    const unsigned int stencilmatrixsize = elements * sizeof(int);
    int * gs_image = (int*)malloc(stencilmatrixsize);
    memset(gs_image, 0, stencilmatrixsize);

    if (ascii) {
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                int offset = tile_width + kw + kw/2 + (i*(cols+kw));
                gs_image[i + offset] = input_image_int[i];
            }
        }
    } else {
        int stencil = kw/2;
        for (int i = 0; i < rows+kw; i++) {
            for (int j = 0; j < cols+kw; j++) {
                if (i < (kw/2) || i >= rows || j < (kw/2) || j >= cols){
                    gs_image[(i*(rows+kw)) + j] = 0;
                } else {
                    int offset = ((i-stencil)*cols) + (j-stencil);
                    gs_image[(i*(rows+kw))+ j] = input_image_char[offset] - '0';
                }
            }
        }
    }
    int *d_gs_image;
    int *d_gs_image_result;
    hipMalloc((int**)&d_gs_image, stencilmatrixsize);
    hipMalloc((int**)&d_gs_image_result, stencilmatrixsize);
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipMemcpy(d_gs_image, gs_image, stencilmatrixsize, hipMemcpyHostToDevice);
    //gpuErrchk(cudaPeekAtLastError());
    //gpuErrchk(cudaDeviceSynchronize());
    int smem_size = (tile_width + kw) * (tile_width + kw) * sizeof(int) * 2;
    hipEventRecord(initstop);
    hipEventSynchronize(initstop);
    float initmilliseconds = 0;
    hipEventElapsedTime(&initmilliseconds, initstart, initstop);
    if (true) {
        if (output) {
            std::ofstream outputFile;
            outputFile.open(file, std::ios_base::app);
            outputFile << "" << initmilliseconds/1000 << ";";
            printf("%.2f", initmilliseconds/1000);
            outputFile.close();
        }
    }
 
   hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    for (int run = 0; run < iterations; ++run) {
        // TODO make multiple GPUs
        // TODO SM Variant
        if(!shared_mem){
            dim3 dimBlock(1024);
            dim3 dimGrid((rows*cols) / dimBlock.x);
            calcGaussian<<<dimGrid, dimBlock, smem_size, stream1>>>(d_gs_image, d_gs_image_result, cols, kw);
            calcGaussian<<<dimGrid, dimBlock, smem_size, stream1>>>(d_gs_image_result, d_gs_image, cols, kw);
        } else{
            if (tile_width == 32) {
                dim3 dimBlock(tile_width, tile_width);
                dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x,
                             (cols + dimBlock.y - 1) / dimBlock.y);
                calcGaussian_fixed_SM<<<dimGrid, dimBlock, smem_size, stream1>>>(d_gs_image, d_gs_image_result, cols, rows, kw, tile_width);
                calcGaussian_fixed_SM<<<dimGrid, dimBlock, smem_size, stream1>>>(d_gs_image_result, d_gs_image, cols, rows, kw, tile_width);
            } else {
                dim3 dimBlock(tile_width, tile_width);
                dim3 dimGrid((rows + dimBlock.x - 1) / dimBlock.x,
                             (cols + dimBlock.y - 1) / dimBlock.y);
                calcGaussianSM<<<dimGrid, dimBlock, smem_size, stream1>>>(d_gs_image, d_gs_image_result, cols, rows, kw, tile_width);
                calcGaussianSM<<<dimGrid, dimBlock, smem_size, stream1>>>(d_gs_image_result, d_gs_image, cols, rows, kw, tile_width);
            }
        }
        if (DEBUG) {
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
        }
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    if (true) {
        if (output) {
            std::ofstream outputFile;
            outputFile.open(file, std::ios_base::app);
            outputFile << "" << milliseconds/1000 << ";";
            printf("%.2f", milliseconds/1000);
            outputFile.close();
        }
    }
    // TODO COPY BACK
    hipMemcpy(gs_image, d_gs_image_result, stencilmatrixsize, hipMemcpyDeviceToHost);

    writePGM(out_file, gs_image, rows+kw, cols+kw, max_color);
    return 0;
}

int init(int row, int col)
{
    if (ascii) return input_image_int[row*cols+col];
    else return input_image_char[row*cols+col];
}
int main(int argc, char **argv) {
    std::cout << "\n\n************* Starting the Gaussian Blur *************\n ";

    int nGPUs = 1;
    int nRuns = 1;
    int iterations = MAX_ITER;
    int tile_width = DEFAULT_TILE_WIDTH;
    float cpu_fraction = 0.0;
    //bool warmup = false;
    bool output = false;
    bool shared_mem = false;
    int kw = 2;
    std::string in_file, out_file, file, nextfile; //int kw = 10;
    file = "result_travel.csv";
    if (argc >= 8) {
        nGPUs = atoi(argv[1]);
        nRuns = atoi(argv[2]);
        cpu_fraction = atof(argv[3]);
        if (cpu_fraction > 1) {
            cpu_fraction = 1;
        }
        tile_width = atoi(argv[4]);
        iterations = atoi(argv[5]);
        if (atoi(argv[6]) == 1) {
            shared_mem = true;
        }
        kw = atoi(argv[7]);

    }
    std::string shared = shared_mem ? "SM" : "GM";

    if (argc == 9) {
        in_file = argv[9];
        size_t pos = in_file.find(".");
        out_file = in_file;
        std::stringstream ss;
        ss << "_" << nGPUs << "_" << iterations << "_" << shared <<  "_" << tile_width << "_" << kw << "_gaussian";
        out_file.insert(pos, ss.str());
    } else {
        in_file = "travelsquaresquare.pgm";
        std::stringstream oo;
        oo << in_file << "_" << nGPUs << "_" << iterations << "_" << shared <<  "_" << tile_width << "_" << kw << "_gaussian.pgm";
        out_file = oo.str();
    }
    output = true;
    std::stringstream ss;
    ss << file << "_" << iterations;
    nextfile = ss.str();

    int iterations_used = 0;
    for (int r = 0; r < nRuns; ++r) {
        testGaussian(in_file, out_file, output, tile_width, iterations, iterations_used, nextfile, shared_mem, kw);
    }

    if (output) {
        std::ofstream outputFile;
        outputFile.open(nextfile, std::ios_base::app);
        outputFile << "" + std::to_string(nGPUs) + ";" + std::to_string(tile_width) +";" + std::to_string(iterations) + ";" +
        std::to_string(iterations_used) + ";\n";
        outputFile.close();
    }
    std::cout << "\n************* Finished the Gaussian Blur *************\n ";

    return 0;
}